// Compilar con:
// nvcc mMultiplication.cu -o mMultiplication
// ./mMultiplication
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 4

typedef struct {
  int width;
  int height;
  float *elements;
} Matrix;

__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C) {
  int row = threadIdx.y;
  int col = threadIdx.x;
  float value = 0;

  for (int k = 0; k < A.width; ++k) {
    float a = A.elements[row * A.width + k];
    float b = B.elements[k * B.width + col];
    value += a * b;
  }

  C.elements[row * C.width + col] = value;
}

int main() {
  float h_A[N * N] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16};
  float h_B[N * N] = {16, 15, 14, 13, 12, 11, 10, 9, 8, 7, 6, 5, 4, 3, 2, 1};
  float h_C[N * N];

  float *d_A, *d_B, *d_C;
  size_t size = N * N * sizeof(float);
  hipMalloc(&d_A, size);
  hipMalloc(&d_B, size);
  hipMalloc(&d_C, size);

  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  Matrix A = {N, N, d_A};
  Matrix B = {N, N, d_B};
  Matrix C = {N, N, d_C};

  dim3 dimBlock(N, N);
  MatMulKernel<<<1, dimBlock>>>(A, B, C);

  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  printf("\nMatriz Resultante C:\n");
  for (int i = 0; i < N; ++i) {
    for (int j = 0; j < N; ++j) {
      printf("%6.1f ", h_C[i * N + j]);
    }
    printf("\n");
  }
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  return 0;
}
